#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <fenv.h>
#include <stdio.h>
#include <sys/time.h>

double getTime();
void handleErrors(hipError_t);


int main(int argc, char *argv[]) {
	if (argc != 3) { printf("usage: ./fft_cuda DEVICE MATRIX_SIZE NUM_THREADS\n"); exit(1); }


	hipfftHandle plan;
	hipfftComplex *data, *dev_data;

	double totalTime, startTime;
	totalTime = 0;

	int deviceNum = atoi(argv[1]);
	int matrixSize = atoi(argv[2]);
	int numThreads = atoi(argv[3]);

	int n[2] = {matrixSize, matrixSize};

	char x = ((hipSetDevice(deviceNum))== hipSuccess)? 'Y' : 'N';
	
	/* Initialize the sequence. */
	srand(654);
	data = (hipfftComplex *) malloc(sizeof(hipfftComplex) * matrixSize * matrixSize);
	int i, j;
    // double pdata=0;
    for (i = 0; i < matrixSize; ++i) {
        for (j = 0; j < matrixSize; ++j) {
          data[i*matrixSize+j][0] = i; 
          data[i*matrixSize+j][1] = 0;
          // pdata += data[i*matrixSize+j][0] * data[i*matrixSize+j][0] + data[i*matrixSize+j][1] * data[i*matrixSize+j][1];
        }
    }

	/* Create a 2D FFT plan. */
	if (hipfftPlanMany(&plan, 2, n,
					  NULL, 1, 0,
					  NULL, 1, 0,
					  HIPFFT_C2C,BATCH) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT Error: Unable to create plan\n");
		return;	
	}

	if (cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_NATIVE)!= HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT Error: Unable to set compatibility mode to native\n");
		return;		
	}

	hipMalloc((void **)&dev_data, sizeof(hipfftComplex) * matrixSize * matrixSize);	
	hipMemcpy(dev_data, &data, sizeof(hipfftComplex) * matrixSize * matrixSize, hipMemcpyHostToDevice);

	// numThreads can range from 1..7 => {1,4,16,64,256,512,1024}
	int bH, bW, t;
	if (numThreads <= 2) {
		bH = (int) pow(4, numThreads-1);
		bW = 1;
	} else if (numThreads <= 4) {
		bH = bW = 4*(numThreads-2);
	} else {
		bH = bW = 16;
		t = (int) pow(2, numThreads - 5);
	}

	dim3 blocks(bH, bW, 1);
	dim3 threads(t, 1, 1);

	startTime = getTime();
	hipfftExecC2R(plan, data, data);
	hipDeviceSynchronize();
	totalTime += getTime() - startTime;
	handleErrors(hipGetLastError());

	hipfftDestroy(plan);
	hipFree(data);

	printf("%.5f\n", totalTime);

	return 0;
}


double getTime() {
	timeval thetime;
	gettimeofday( &thetime, 0 );
	return thetime.tv_sec + thetime.tv_usec / 1000000.0;
}


void handleErrors(hipError_t err) {
	if (err != hipSuccess) {
		printf("Kernel: %s\n",hipGetErrorString(err));
	}
}