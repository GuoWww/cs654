#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <math.h>
#include <fenv.h>
#include <stdio.h>
#include <sys/time.h>

double getTime();
void handleErrors(hipError_t);


int main(int argc, char *argv[]) {
	if (argc != 3) { printf("usage: ./fft_cuda DEVICE MATRIX_SIZE\n"); exit(1); }

	hipfftHandle plan;
	hipfftComplex *data, *dev_data;

	double totalTime, startTime;
	totalTime = 0;

	int deviceNum = atoi(argv[1]);
	int matrixSize = atoi(argv[2]);

	char x = ((hipSetDevice(deviceNum))== hipSuccess)? 'Y' : 'N';
	
	/* Initialize the sequence. */
	srand(654);
	data = (hipfftComplex *) malloc(sizeof(hipfftComplex) * matrixSize * matrixSize);
	int i, j;
	// double pdata=0;
	for (i = 0; i < matrixSize; ++i) {
		for (j = 0; j < matrixSize; ++j) {
			data[i*matrixSize+j].x = i;
			data[i*matrixSize+j].y = 0;
			// pdata += data[i*matrixSize+j][0] * data[i*matrixSize+j][0] + data[i*matrixSize+j][1] * data[i*matrixSize+j][1];
		}
	}


	/* Create a 2D FFT plan. */
	if (hipfftPlan2d(&plan, matrixSize, matrixSize, HIPFFT_C2C) != HIPFFT_SUCCESS) {
		fprintf(stderr, "CUFFT Error: Unable to create plan\n");
		return 0;	
	}

	if (cufftSetCompatibilityMode(plan, CUFFT_COMPATIBILITY_NATIVE)!= HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT Error: Unable to set compatibility mode to native\n");
		return 0;		
	}

	hipMalloc((void **)&dev_data, sizeof(hipfftComplex) * matrixSize * matrixSize);	
	hipMemcpy(dev_data, data, sizeof(hipfftComplex) * matrixSize * matrixSize, hipMemcpyHostToDevice);

	startTime = getTime();
	hipfftExecC2C(plan, data, data, HIPFFT_FORWARD);
	hipDeviceSynchronize();
	totalTime += getTime() - startTime;
	handleErrors(hipGetLastError());

	hipfftDestroy(plan);
	hipFree(data);

	printf("%.5f\n", totalTime);

	return 0;
}


double getTime() {
	timeval thetime;
	gettimeofday( &thetime, 0 );
	return thetime.tv_sec + thetime.tv_usec / 1000000.0;
}


void handleErrors(hipError_t err) {
	if (err != hipSuccess) {
		printf("Kernel: %s\n",hipGetErrorString(err));
	}
}
